#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <math.h>

#include <stdio.h>
#include "CudaProto.h"

#include <iostream>

using namespace std;

// Functions signatures
__global__ void initDistancesProductsToClusters(double* dev_distancesProductsToClusters, const int numOfClusters);
__global__ void calcSubtractionsPower(const double* dev_products, const double* dev_clusters, double* centersOfClustersPerProduct, const int numOfClusters, const int productSize);
__global__ void calcSumOfSubtractionsPower(double* dev_distancesProductsToClusters, const double* dev_centersOfClustersPerProduct, const int numOfClusters, const int productSize);
__global__ void doSquareRoot(double* dev_distancesProductsToClusters, const int numOfClusters);
void print(double* arr, int size);
void free(double*& dev_products, double*& dev_clusters);

hipError_t calcDistancesFromClusters(const int numOfProducts, const int productSize, const int numOfClusters, 
													const double* products, double* clusters, double* distancesProductsToClusters)
{
	double* dev_products; // Array of all the values of all the products
	double* dev_clusters; // Array of all the values of all the clusters' centers
	double* dev_centersOfClustersPerProduct; // Array of all the clusters per product (for all the products)
	double* dev_distancesProductsToClusters; // Array of all distances from each product to each cluster's center

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		free(dev_products, dev_clusters);
	}

	// Allocate GPU buffers for products
	cudaStatus = hipMalloc((void**)&dev_products, numOfProducts * productSize * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc products failed!");
		free(dev_products, dev_clusters);
	}

	// Allocate GPU buffers for clusters
	cudaStatus = hipMalloc((void**)&dev_clusters, numOfClusters * productSize * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc clusters failed!");
		free(dev_products, dev_clusters);
	}

	// Allocate GPU buffers for centers of clusters per product
	cudaStatus = hipMalloc((void**)&dev_centersOfClustersPerProduct, productSize * numOfProducts * numOfClusters * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc centers of clusters per product failed!");
		free(dev_products, dev_clusters);
	}

	// Allocate GPU buffers for distance from products to clusters centers
	cudaStatus = hipMalloc((void**)&dev_distancesProductsToClusters, numOfProducts * numOfClusters * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc distance from products to clusters centers failed!");
		free(dev_products, dev_clusters);
	}

	// Copy input of the products from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_products, products, numOfProducts * productSize * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy products failed!");
		free(dev_products, dev_clusters);
	}

	// Copy input of the clusters from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_clusters, clusters, numOfClusters * productSize * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy clusters failed!");
		free(dev_products, dev_clusters);
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching initExtendHistograma!\n", cudaStatus);
		free(dev_products, dev_clusters);
	}
	
	// Init the array of distances from each product to each cluster's center
	initDistancesProductsToClusters << <numOfProducts, numOfClusters >> > (dev_distancesProductsToClusters, numOfClusters);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "initDistancesProductsToClusters launch failed: %s\n", hipGetErrorString(cudaStatus));
		free(dev_products, dev_clusters);
	}

	// For each product, calc the subtraction power of each value of the product and the value at the same index of each cluster's center
	// (product[i] - clusterCenter[i])^2 --> do this calculation for all the clusters' centers
	calcSubtractionsPower << < numOfProducts, productSize >> > (dev_products, dev_clusters, dev_centersOfClustersPerProduct, numOfClusters, productSize);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "calcSubtractionsPower launch failed: %s\n", hipGetErrorString(cudaStatus));
		free(dev_products, dev_clusters);
	}

	// Calc the sum of all the subtracions power
	calcSumOfSubtractionsPower << < numOfProducts, numOfClusters >> > (dev_distancesProductsToClusters, dev_centersOfClustersPerProduct, numOfClusters, productSize);	

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "calcSumOfSubtractionsPower launch failed: %s\n", hipGetErrorString(cudaStatus));
		free(dev_products, dev_clusters);
	}

	// Do square root on each sum of subtraction power (to finish the calculation of distance from each product to each cluster's center)
	doSquareRoot << <numOfProducts, numOfClusters >> > (dev_distancesProductsToClusters, numOfClusters);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "doSquareRoot launch failed: %s\n", hipGetErrorString(cudaStatus));
		free(dev_products, dev_clusters);
	}

	// Copy the distances of each product from each cluster's center from GPU buffer to host memory.
	cudaStatus = hipMemcpy(distancesProductsToClusters, dev_distancesProductsToClusters, numOfProducts * numOfClusters * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy distances of each product from each cluster's center failed!");
		free(dev_products, dev_clusters);
	}

	return cudaStatus;
}


// Functions
__global__ void initDistancesProductsToClusters(double* dev_distancesProductsToClusters, const int numOfClusters)
{
	int productIndex = blockIdx.x;
	int clusterIndex = threadIdx.x;

	dev_distancesProductsToClusters[productIndex * numOfClusters + clusterIndex] = 0;
}

__global__ void calcSubtractionsPower(const double* dev_products, const double* dev_clusters, double* centersOfClustersPerProduct, const int numOfClusters, const int productSize)
{
	int productIndex = blockIdx.x;
	int valueIndex = threadIdx.x;

	for (int clusterIndex = 0; clusterIndex < numOfClusters; clusterIndex++)
	{
		centersOfClustersPerProduct[productIndex * productSize * numOfClusters + clusterIndex * productSize + valueIndex] =
			pow(dev_products[productIndex * productSize + valueIndex] - dev_clusters[clusterIndex * productSize + valueIndex], 2);
	}
}

__global__ void calcSumOfSubtractionsPower(double* dev_distancesProductsToClusters, const double* dev_centersOfClustersPerProduct,
	const int numOfClusters, const int productSize)
{
	int productIndex = blockIdx.x;
	int clusterIndex = threadIdx.x;;

	for (int valueIndex = 0; valueIndex < productSize; valueIndex++)
	{
		dev_distancesProductsToClusters[productIndex * numOfClusters + clusterIndex] += 
			dev_centersOfClustersPerProduct[productIndex * productSize * numOfClusters + productSize * clusterIndex + valueIndex];
	}
}

__global__ void doSquareRoot(double* dev_distancesProductsToClusters, const int numOfClusters)
{
	int productIndex = blockIdx.x;
	int clusterIndex = threadIdx.x;

	dev_distancesProductsToClusters[productIndex * numOfClusters + clusterIndex] = sqrt(dev_distancesProductsToClusters[productIndex * numOfClusters + clusterIndex]);
}

void free(double*& dev_products, double*& dev_clusters)
{
	hipFree(dev_products);
	hipFree(dev_clusters);
}

void print(double* arr, int size)
{
	for (int i = 0; i < size; i++)
	{
		printf("%d ", arr[i]);
	}
}